/*****************************************************************************
 * File:        matrix_add_managed.cu
 * Description: Matrix addition example using CUDA mamanged memory (unified memory).
 *              CUDA will automatically manage the transfer of data to and from
 *              the GPU as needed by the application. There is no need for the
 *              programmer to use cudaMemcpy, cudaHostGetDevicePointer, or any other
 *              CUDA API.
 *              
 * Compile:     nvcc -O3 matrix_add_managed.cu -o managed
 * Run:         ./managed <n>
 *                  <n> : specify matrix dimension as (2^n, 2^n)
 *****************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_ERROR_CHECK(err) \
    if (err != hipError_t::hipSuccess) { \
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__); \
        exit(err); \
    }

void initMatrix(float* in, int const num_elements)
{
    for (int i = 0; i < num_elements; i++) {
        in[i] = rand() / (float)RAND_MAX;
    }
}

void matrixAddOnHost(float const* A, float const* B, float* C, int const nx, int const ny)
{
    float const* ia = A;
    float const* ib = B;
    float* ic = C;

    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            ic[ix] = ia[ix] + ib[ix];
        }

        ia += nx;
        ib += nx;
        ic += nx;
    }
}

void checkResult(float const* host_ref, float const* gpu_ref, int const num_elements)
{
    for (int i = 0; i < num_elements; i++) {
        if (host_ref[i] != gpu_ref[i]) {
            printf("different on %dth element, host: %f / gpu: %f\n", i, host_ref[i], gpu_ref[i]);
            break;
        }
    }
}

__global__
void warmup(float const* A, float const* B, float* C, int const nx, int const ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__
void matrixAddGPU(float const* A, float const* B, float* C, int const nx, int const ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny) {
        C[idx] = A[idx] + B[idx];
    }
}

int main(int argc, char** argv)
{
    // setup device
    int dev = 0;
    hipDeviceProp_t prop;
    CUDA_ERROR_CHECK(hipGetDeviceProperties(&prop, dev));
    printf("> Matrix Addition(Manual) at device %d: %s\n", dev, prop.name);
    CUDA_ERROR_CHECK(hipSetDevice(dev));

    // setup matrix dimensions (2048 x 2048)
    int pow = 12;
    if (argc > 1) pow = atoi(argv[1]);
    int nx = 1 << pow;
    int ny = 1 << pow;

    printf("> with matrix %d x %d\n", nx, ny);
    size_t bytes = nx * ny * sizeof(float);

    // CUDA event to estimate elapsed time
    hipEvent_t start, end;
    float msec = 0.f;
    CUDA_ERROR_CHECK(hipEventCreate(&start));
    CUDA_ERROR_CHECK(hipEventCreate(&end));

    // malloc host memory
    float *A, *B, *host_ref, *gpu_ref;
    CUDA_ERROR_CHECK(hipMallocManaged(&A, bytes));
    CUDA_ERROR_CHECK(hipMallocManaged(&B, bytes));
    CUDA_ERROR_CHECK(hipMallocManaged(&host_ref, bytes));
    CUDA_ERROR_CHECK(hipMallocManaged(&gpu_ref, bytes));

    // initialize data at host side
    CUDA_ERROR_CHECK(hipEventRecord(start));
    initMatrix(A, nx * ny);
    initMatrix(B, nx * ny);
    CUDA_ERROR_CHECK(hipEventRecord(end));
    CUDA_ERROR_CHECK(hipEventSynchronize(end));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));
    printf("initialization: \t %f sec\n", msec * 1e-3);

    memset(host_ref, 0, bytes);
    memset(gpu_ref, 0, bytes);

    // matrix addition at host side for checking result
    CUDA_ERROR_CHECK(hipEventRecord(start));
    matrixAddOnHost(A, B, host_ref, nx, ny);
    CUDA_ERROR_CHECK(hipEventRecord(end));
    CUDA_ERROR_CHECK(hipEventSynchronize(end));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));
    printf("matrixAdd on host:\t %f sec\n", msec * 1e-3);


    // setup execution configuration
    int blockx = 32;
    int blocky = 32;
    dim3 block(blockx, blocky);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // warm-up kernel with unified memory all pages will migrate from host to device
    warmup<<<grid, block>>>(A, B, gpu_ref, nx, ny);

    // kernel launch
    CUDA_ERROR_CHECK(hipEventRecord(start));
    matrixAddGPU<<<grid, block>>>(A, B, gpu_ref, nx, ny);
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipEventRecord(end));
    CUDA_ERROR_CHECK(hipEventSynchronize(end));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));
    printf("matrixAdd on gpu :\t %f sec <<<(%d,%d), (%d,%d)>>>\n", msec * 1e-3, grid.x, grid.y, block.x, block.y);

    // check kernel error
    CUDA_ERROR_CHECK(hipGetLastError());

    // check results
    checkResult(host_ref, gpu_ref, nx * ny);

    // free host & device memory
    CUDA_ERROR_CHECK(hipFree(A));
    CUDA_ERROR_CHECK(hipFree(B));
    CUDA_ERROR_CHECK(hipFree(host_ref));
    CUDA_ERROR_CHECK(hipFree(gpu_ref));

    CUDA_ERROR_CHECK(hipDeviceReset());
    return 0;
}