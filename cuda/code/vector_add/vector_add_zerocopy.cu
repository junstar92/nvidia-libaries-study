/*****************************************************************************
 * File:        vector_add_zerocopy.cu
 * Description: Parallel Vector Addition with Zero-copy Memory. A + B = C
 *              
 * Compile:     nvcc -O3 -o vector_add_zerocopy vector_add_zerocopy.cu
 * Run:         ./vector_add <n>
 *                  <n> : the number of vector elements = n power(s) of 2
 *****************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CUDA_ERROR_CHECK(err) \
    if (err != hipError_t::hipSuccess) { \
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__); \
        exit(err); \
    }

void initVector(float* vec, const int num_elements)
{
    for (int i = 0; i < num_elements; i++) {
        vec[i] = rand() / (float)RAND_MAX;
    }
}

void vectorAddOnHost(float const* a, float const* b, float* c, const int num_elements)
{
    for (int i = 0; i < num_elements; i++) {
        c[i] = a[i] + b[i];
    }
}

__global__
void vectorAddKernel(float const* a, float const* b, float* c, int const num_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < num_elements)
        c[i] = a[i] + b[i];
}

void checkResult(float const* host_ref, float const* gpu_ref, int const num_elements)
{
    for (int i = 0; i < num_elements; i++) {
        if (host_ref[i] != gpu_ref[i]) {
            printf("Result verification failed at element %d (%f != %f)\n", i, host_ref[i], gpu_ref[i]);
            return;
        }
    }
}

int main(int argc, char** argv)
{
    int pow = 22;
    if (argc > 1) pow = strtol(argv[1], NULL, 10);
    int num_elements = 1 << pow;
    size_t bytes = num_elements * sizeof(float);

    if (pow < 18) {
        printf("> Vector size: %d elements bytes: %3.0f KB\n", num_elements, (float)bytes / (1024.f));
    }
    else {
        printf("> Vector size: %d elements bytes: %3.0f MB\n", num_elements, (float)bytes / (1024.f*1024.f));
    }

    hipEvent_t start, stop;
    float msec, total = 0.f;
    CUDA_ERROR_CHECK(hipEventCreate(&start));
    CUDA_ERROR_CHECK(hipEventCreate(&stop));

    /*************** Case 1: using device memory ***************/
    // allocate the host memory
    float *h_a, *h_b, *host_ref, *gpu_ref;
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    host_ref = (float*)malloc(bytes);
    gpu_ref = (float*)malloc(bytes);

    // init vector a, b
    initVector(h_a, num_elements);
    initVector(h_b, num_elements);
    (void*)memset(host_ref, 0, bytes);
    (void*)memset(gpu_ref, 0, bytes);
    
    // add vector at host side for result check
    vectorAddOnHost(h_a, h_b, host_ref, num_elements);

    // malloc device global memory
    float *d_a, *d_b, *d_c;
    CUDA_ERROR_CHECK(hipMalloc(&d_a, bytes));
    CUDA_ERROR_CHECK(hipMalloc(&d_b, bytes));
    CUDA_ERROR_CHECK(hipMalloc(&d_c, bytes));

    // transfer data from host to device
    CUDA_ERROR_CHECK(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice));

    // setup execution configuration
    int threads_per_block = 512;
    int blocks_per_grid = (num_elements + threads_per_block - 1) / threads_per_block;

    // launch kernel at host side
    for (int i = 0; i < 100; i++) {
        CUDA_ERROR_CHECK(hipEventRecord(start));
        vectorAddKernel<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, num_elements);
        CUDA_ERROR_CHECK(hipEventRecord(stop));
        CUDA_ERROR_CHECK(hipEventSynchronize(stop));
        CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, stop));
        total += msec;
    }

    // copy kernel result back to host side
    CUDA_ERROR_CHECK(hipMemcpy(gpu_ref, d_c, bytes, hipMemcpyDeviceToHost));

    // check device results
    checkResult(host_ref, gpu_ref, num_elements);

    // free device global memory
    CUDA_ERROR_CHECK(hipFree(d_a));
    CUDA_ERROR_CHECK(hipFree(d_b));
    free(h_a);
    free(h_b);

    printf("> vectorAddKernel(global memory)    Elapsed Time: %f ms\n", total / 100);

    /*************** Case 2: using zero-copy memory ***************/
    total = 0.f;
    // allocate zero-copy memory
    unsigned int flags = hipHostMallocMapped;
    CUDA_ERROR_CHECK(hipHostAlloc(&h_a, bytes, flags));
    CUDA_ERROR_CHECK(hipHostAlloc(&h_b, bytes, flags));

    // initialize data at host side
    initVector(h_a, num_elements);
    initVector(h_b, num_elements);
    (void*)memset(host_ref, 0, bytes);
    (void*)memset(gpu_ref, 0, bytes);
    
    // add vector at host side for result check
    vectorAddOnHost(h_a, h_b, host_ref, num_elements);

    // pass the pointer to device
    CUDA_ERROR_CHECK(hipHostGetDevicePointer(&d_a, h_a, 0));
    CUDA_ERROR_CHECK(hipHostGetDevicePointer(&d_b, h_b, 0));

    // launch kernel with zero-copy memory
    for (int i = 0; i < 100; i++) {
        CUDA_ERROR_CHECK(hipEventRecord(start));
        vectorAddKernel<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, num_elements);
        CUDA_ERROR_CHECK(hipEventRecord(stop));
        CUDA_ERROR_CHECK(hipEventSynchronize(stop));
        CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, stop));
        total += msec;
    }

    // copy kernel result back to host side
    CUDA_ERROR_CHECK(hipMemcpy(gpu_ref, d_c, bytes, hipMemcpyDeviceToHost));

    // check device results
    checkResult(host_ref, gpu_ref, num_elements);

    // free memory
    CUDA_ERROR_CHECK(hipFree(d_c));
    CUDA_ERROR_CHECK(hipHostFree(h_a));
    CUDA_ERROR_CHECK(hipHostFree(h_b));
    free(host_ref);
    free(gpu_ref);

    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, stop));
    printf("> vectorAddKernel(zero-copy memory) Elapsed Time: %f ms\n", total / 100);

    hipDeviceReset();
    printf("Done\n");

    return 0;
}