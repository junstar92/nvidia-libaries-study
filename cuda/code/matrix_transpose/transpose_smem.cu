/*****************************************************************************
 * File:        transpose_smem.cu
 * Description: Implement various kernels for transposing a matrix using a variety
 *              of optimizations, including shared memory, unrolling, and
 *              memory padding
 *              
 * Compile:     nvcc -O3 -o transpose_smem transpose_smem.cu
 * Run:         ./transpose_smem
 *****************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_ERROR_CHECK(err) \
    if (err != hipError_t::hipSuccess) { \
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__); \
        exit(err); \
    }

#define BDIMX 32
#define BDIMY 16
#define PADDING 2

void initMatrix(float* in, int const num_elements)
{
    for (int i = 0; i < num_elements; i++) {
        in[i] = rand() / (float)RAND_MAX;
    }
}

void transposeHost(float* out, float const* in, int const nx, int const ny)
{
    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            out[ix * ny + iy] = in[iy * nx + ix];
        }
    }
}

void checkResult(float const* host_ref, float const* gpu_ref, int const num_elements)
{
    double eps = 1e-8;
    
    for (int i = 0; i < num_elements; i++) {
        if (host_ref[i] != gpu_ref[i]) {
            printf("different on %dth element, host: %f / gpu: %f\n", i, host_ref[i], gpu_ref[i]);
            break;
        }
    }
}

__global__
void warmup(float* out, float const* in, int const nx, int const ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny) {
        out[iy * nx + ix] = in[iy * nx + ix];
    }
}

// 0 copy kernel: read by rows, write by rows using global memory
__global__
void copyRow(float* out, float const* in, int const nx, int const ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny) {
        out[iy * nx + ix] = in[iy * nx + ix];
    }
}

// 1 copy kernel: read by columns, write by columns using global memory
__global__
void copyCol(float* out, float const* in, int const nx, int const ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny) {
        out[ix * ny + iy] = in[ix * ny + iy];
    }
}

// 2 transpose kernel: read by rows, write by columns using global memory
__global__
void transposeNaiveRow(float* out, float const* in, int const nx, int const ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny) {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}

// 3 transpose kernel: read by columns, write by rows using global memory
__global__
void transposeNaiveCol(float* out, float const* in, int const nx, int const ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny) {
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}

// 4 transpose kernel: read by rows, write by columns + unroll 4 blocks using global memory
__global__
void transposeUnroll4Row(float* out, float const* in, int const nx, int const ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    if (ix + 3 * blockDim.x < nx && iy < ny) {
        out[to] = in [ti];
        out[to + ny * blockDim.x] = in[ti + blockDim.x];
        out[to + ny * 2 * blockDim.x] = in[ti + 2 * blockDim.x];
        out[to + ny * 3 * blockDim.x] = in[ti + 3 * blockDim.x];
    }
}

// 5 transpose kernel: read by columns, write by rows + unroll 4 blocks using global memory
__global__
void transposeUnroll4Col(float* out, float const* in, int const nx, int const ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    if (ix + 3 * blockDim.x < nx && iy < ny) {
        out[ti] = in [to];
        out[ti + blockDim.x] = in[to + blockDim.x * ny];
        out[ti + 2 * blockDim.x] = in[to + 2 * blockDim.x * ny];
        out[ti + 3 * blockDim.x] = in[to + 3 * blockDim.x * ny];
    }
}

// 6 transpose kernel: read by rows, write by columns + using shared memory
__global__
void transposeSmem(float* out, float const* in, int const nx, int const ny)
{
    // static shared memory
    __shared__ float tile[BDIMY][BDIMX];

    // coordinate in original matrix
    unsigned int ix, iy, ti, to;
    ix = blockDim.x * blockIdx.x + threadIdx.x;
    iy = blockDim.y * blockIdx.y + threadIdx.y;

    // linear global memory index for original matrix
    ti = iy * nx + ix;

    // thread index in transposed block
    unsigned int bidx, irow, icol;
    bidx = blockDim.x * threadIdx.y + threadIdx.x;
    irow = bidx / blockDim.y;
    icol = bidx % blockDim.y;

    // coordinate in transposed matrix
    ix = blockDim.y * blockIdx.y + icol;
    iy = blockDim.x * blockIdx.x + irow;

    // linear global memory index for transposed matrix
    to = iy * ny + ix;

    if (ix < nx && iy < ny) {
        // load data from global memory to shared memory
        tile[threadIdx.y][threadIdx.x] = in[ti];
        __syncthreads();

        // store data to global memory from shared memory
        out[to] = tile[icol][irow];
    }
}

// 7 transpose kernel: read by rows, write by columns with memory padding + using shared memory
__global__
void transposeSmemPad(float* out, float const* in, int const nx, int const ny)
{
    // static shared memory
    __shared__ float tile[BDIMY][BDIMX + PADDING];

    // coordinate in original matrix
    unsigned int ix, iy, ti, to;
    ix = blockDim.x * blockIdx.x + threadIdx.x;
    iy = blockDim.y * blockIdx.y + threadIdx.y;

    // linear global memory index for original matrix
    ti = iy * nx + ix;

    // thread index in transposed block
    unsigned int bidx, irow, icol;
    bidx = blockDim.x * threadIdx.y + threadIdx.x;
    irow = bidx / blockDim.y;
    icol = bidx % blockDim.y;

    // coordinate in transposed matrix
    ix = blockDim.y * blockIdx.y + icol;
    iy = blockDim.x * blockIdx.x + irow;

    // linear global memory index for transposed matrix
    to = iy * ny + ix;

    if (ix < nx && iy < ny) {
        // load data from global memory to shared memory
        tile[threadIdx.y][threadIdx.x] = in[ti];
        __syncthreads();

        // store data to global memory from shared memory
        out[to] = tile[icol][irow];
    }
}

// 8 transpose kernel: read by rows, write by columns + using shared memory + unrolling 2 thead blocks
__global__
void transposeSmemUnroll2(float* out, float const* in, int const nx, int const ny)
{
    // static 1D shared memory
    __shared__ float tile[BDIMY * BDIMX * 2];

    // coordinate in original matrix
    unsigned int ix, iy, ti, to, ix2, iy2;
    ix = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    iy = blockDim.y * blockIdx.y + threadIdx.y;

    // linear global memory index for original matrix
    ti = iy * nx + ix;

    // thread index in transposed block
    unsigned int bidx, irow, icol;
    bidx = blockDim.x * threadIdx.y + threadIdx.x;
    irow = bidx / blockDim.y;
    icol = bidx % blockDim.y;

    // coordinate in transposed matrix
    ix2 = blockDim.y * blockIdx.y + icol;
    iy2 = 2 * blockDim.x * blockIdx.x + irow;

    // linear global memory index for transposed matrix
    to = iy2 * ny + ix2;

    if (ix + blockDim.x < nx && iy < ny) {
        // load two rows from global memory to shared memory
        unsigned int row_idx = 2 * blockDim.x * threadIdx.y + threadIdx.x;
        tile[row_idx] = in[ti];
        tile[row_idx + BDIMX] = in[ti + BDIMX];
        __syncthreads();

        // store two rows to global memory from two columns of shared memory
        unsigned int col_idx = 2 * blockDim.x * icol + irow;
        out[to] = tile[col_idx];
        out[to + ny * BDIMX] = tile[col_idx + BDIMX];
    }
}

// 9 transpose kernel: read by rows, write by columns + using shared memory + unrolling 2 thead blocks + memory padding
__global__
void transposeSmemUnroll2Pad(float* out, float const* in, int const nx, int const ny)
{
    // static 1D shared memory
    __shared__ float tile[BDIMY * (BDIMX * 2 + PADDING)];

    // coordinate in original matrix
    unsigned int ix, iy, ti, to, ix2, iy2;
    ix = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    iy = blockDim.y * blockIdx.y + threadIdx.y;

    // linear global memory index for original matrix
    ti = iy * nx + ix;

    // thread index in transposed block
    unsigned int bidx, irow, icol;
    bidx = blockDim.x * threadIdx.y + threadIdx.x;
    irow = bidx / blockDim.y;
    icol = bidx % blockDim.y;

    // coordinate in transposed matrix
    ix2 = blockDim.y * blockIdx.y + icol;
    iy2 = 2 * blockDim.x * blockIdx.x + irow;

    // linear global memory index for transposed matrix
    to = iy2 * ny + ix2;

    if (ix + blockDim.x < nx && iy < ny) {
        // load two rows from global memory to shared memory
        unsigned int row_idx = (2 * blockDim.x + PADDING) * threadIdx.y + threadIdx.x;
        tile[row_idx] = in[ti];
        tile[row_idx + BDIMX] = in[ti + BDIMX];
        __syncthreads();

        // store two rows to global memory from two columns of shared memory
        unsigned int col_idx = (2 * blockDim.x + PADDING) * icol + irow;
        out[to] = tile[col_idx];
        out[to + ny * BDIMX] = tile[col_idx + BDIMX];
    }
}

int main(int argc, char** argv)
{
    // setup device
    int dev = 0;
    hipDeviceProp_t prop;
    CUDA_ERROR_CHECK(hipGetDeviceProperties(&prop, dev));
    printf("> Matrix transpose at device %d: %s\n", dev, prop.name);
    CUDA_ERROR_CHECK(hipSetDevice(dev));

    // setup matrix dimensions (4096 x 4096)
    int nx = 1 << 12;
    int ny = 1 << 12;

    // setup kernel and block size
    int blockx = BDIMX;
    int blocky = BDIMY;

    printf("> with matrix %d x %d\n", nx, ny);
    size_t bytes = nx * ny * sizeof(float);

    // setup execution configuration
    dim3 block(blockx, blocky);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // allocate host memory
    float* h_A = (float*)malloc(bytes);
    float* host_ref = (float*)malloc(bytes);
    float* gpu_ref = (float*)malloc(bytes);

    // initialize host matrix
    initMatrix(h_A, nx * ny);

    // transpose at host side
    transposeHost(host_ref, h_A, nx, ny);

    // allocate device memory
    float *d_A, *d_B;
    CUDA_ERROR_CHECK(hipMalloc(&d_A, bytes));
    CUDA_ERROR_CHECK(hipMalloc(&d_B, bytes));

    // copy data from host to device
    CUDA_ERROR_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));

    // CUDA event to estimate elapsed time
    hipEvent_t start, end;
    float msec;
    CUDA_ERROR_CHECK(hipEventCreate(&start));
    CUDA_ERROR_CHECK(hipEventCreate(&end));

    // warmup to avoid startup overhead
    warmup<<<grid, block>>>(d_B, d_A, nx, ny);

    // setup kernel
    void(*kernel_func)(float*, float const*, int const, int const);
    char* kernel_name;

    for (int i = 0; i < 10; i++) {
        switch (i) {
        default:
        case 0:
            kernel_func = &copyRow;
            kernel_name = "CopyGmemRow           ";
            break;

        case 1:
            kernel_func = &copyCol;
            kernel_name = "CopyGmemCol           ";
            break;

        case 2:
            kernel_func = &transposeNaiveRow;
            kernel_name = "NaiveGmemRow          ";
            break;

        case 3:
            kernel_func = &transposeNaiveCol;
            kernel_name = "NaiveGmemCol          ";
            break;
        
        case 4:
            grid.x = (nx + block.x * 4 - 1) / (block.x * 4);
            kernel_func = &transposeUnroll4Row;
            kernel_name = "NaiveGmemUnroll4Row   ";
            break;
        
        case 5:
            grid.x = (nx + block.x * 4 - 1) / (block.x * 4);
            kernel_func = &transposeUnroll4Col;
            kernel_name = "NaiveGmemUnroll4Col   ";
            break;
        
        case 6:
            grid.x = (nx + block.x - 1) / block.x;
            kernel_func = &transposeSmem;
            kernel_name = "transposeSmem         ";
            break;
        
        case 7:
            grid.x = (nx + block.x - 1) / block.x;
            kernel_func = &transposeSmemPad;
            kernel_name = "transposeSmemPad      ";
            break;
        
        case 8:
            grid.x = (nx + block.x * 2 - 1) / (block.x * 2);
            kernel_func = &transposeSmemUnroll2;
            kernel_name = "transposeSmemUnroll   ";
            break;
        
        case 9:
            grid.x = (nx + block.x * 2 - 1) / (block.x * 2);
            kernel_func = &transposeSmemUnroll2Pad;
            kernel_name = "transposeSmemUnrollPad";
            break;
        }

        // run kernel
        CUDA_ERROR_CHECK(hipEventRecord(start));
        kernel_func<<<grid, block>>>(d_B, d_A, nx, ny);
        CUDA_ERROR_CHECK(hipEventRecord(end));
        CUDA_ERROR_CHECK(hipEventSynchronize(end));
        CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));

        // calculate effective bandwidth
        float bandwidth = 2 * nx * ny * sizeof(float) / 1e9 / (msec * 1.0e-3);
        printf("%s elapsed time: %f ms <<< grid(%d,%d) block(%d,%d)>>> effective bandwidth: %f GB/s\n",
            kernel_name, msec, grid.x, grid.y, block.x, block.y, bandwidth);
        CUDA_ERROR_CHECK(hipGetLastError());

        // check results
        if (i > 1) {
            CUDA_ERROR_CHECK(hipMemcpy(gpu_ref, d_B, bytes, hipMemcpyDeviceToHost));
            checkResult(host_ref, gpu_ref, nx * ny);
        }
    }

    // free host & device memory
    CUDA_ERROR_CHECK(hipFree(d_A));
    CUDA_ERROR_CHECK(hipFree(d_B));
    free(h_A); free(host_ref); free(gpu_ref);

    CUDA_ERROR_CHECK(hipDeviceReset());
    return 0;
}