
#include <hip/hip_runtime.h>
#include <hipblas.h>

void cublas_sgemm(
    int const m, int const n, int const k,
    float const alpha,
    float const* A, float const* B,
    float const beta,
    float* C,
    hipStream_t stream,
    hipblasHandle_t handle
)
{
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, &alpha, A, k, B, n, &beta, C, n);
}