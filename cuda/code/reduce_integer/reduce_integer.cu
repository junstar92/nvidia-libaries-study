/*****************************************************************************
 * File:        reduce_integer.cu
 * Description: Implement kernel functions for reduction(sum) problem
 *                  - recursiveReduce
 *                  - reduceNeighbored
 *                  - reduceNeighboredLess
 *                  - reduceInterleaved
 *                  - reduceUnrolling2
 *                  - reduceUnrolling4
 *                  - reduceUnrolling8
 *                  - reduceUnrollingWarps8
 *                  - reduceCompleteUnrollWarps8
 *                  - reduceCompleteUnroll (template kernel function)
 *              
 * Compile:     nvcc -o reduce_integer reduce_integer.cu
 * Run:         ./reduce_integer <n>
 *                 <n> : block size (1D). (default: 512)
 *****************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_ERROR_CHECK(err) \
    if (err != hipError_t::hipSuccess) { \
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__); \
        exit(err); \
    }

// recursive implementation of interleaved pair approach
int recursiveReduce(int* data, int const size)
{
    // terminate check
    if (size == 1) return data[0];

    // renew the stride
    int const stride = size / 2;

    // in-place reduction
    for (int i = 0; i < stride; i++) {
        data[i] += data[i + stride];
    }

    // call recursively
    return recursiveReduce(data, stride);
}

// neighbored pair implementation with branch divergence
__global__
void reduceNeighbored(int* g_in, int* g_out, unsigned int const n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // convert global data pointer to the local pointer of this block
    int* in = g_in + blockDim.x * blockIdx.x;
    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0) {
            in[tid] += in[tid + stride];
        }
        // synchronize within block
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0)
        g_out[blockIdx.x] = in[0];
}

// neighbored pair implementation with less divergence
__global__
void reduceNeighboredLess(int* g_in, int* g_out, unsigned int const n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // convert global data pointer to the local pointer of this block
    int* in = g_in + blockDim.x * blockIdx.x;
    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory
    int index;
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        // convert tid into local array index
        index = 2 * stride * tid;

        if (index < blockDim.x)
            in[index] += in[index + stride];

        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0)
        g_out[blockIdx.x] = in[0];
}

// interleaved pair implementation with less divergence
__global__
void reduceInterleaved(int* g_in, int* g_out, unsigned int const n)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // convert global data pointer to the local pointer of this block
    int* in = g_in + blockDim.x * blockIdx.x;
    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride)
            in[tid] += in[tid + stride];
        
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0)
        g_out[blockIdx.x] = in[0];
}

// unrolling 2
__global__
void reduceUnrolling2(int* g_in, int* g_out, unsigned int const n)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 2 + threadIdx.x;
    // convert global data pointer to the local pointer of this block
    int* in = g_in + blockDim.x * blockIdx.x * 2;
    
    // unrolling 2 data blocks
    if (idx + blockDim.x < n)
        g_in[idx] += g_in[idx + blockDim.x];
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            in[tid] += in[tid + stride];
        }

        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0)
        g_out[blockIdx.x] = in[0];
}

// unrolling 4
__global__
void reduceUnrolling4(int* g_in, int* g_out, unsigned int const n)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    // convert global data pointer to the local pointer of this block
    int* in = g_in + blockDim.x * blockIdx.x * 4;
    
    // unrolling 4 data blocks
    if (idx + blockDim.x * 3 < n) {
        int sum = 0;
        sum += g_in[idx + blockDim.x];
        sum += g_in[idx + blockDim.x * 2];
        sum += g_in[idx + blockDim.x * 3];
        g_in[idx] += sum;
    }
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            in[tid] += in[tid + stride];
        }

        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0)
        g_out[blockIdx.x] = in[0];
}

// unrolling 8
__global__
void reduceUnrolling8(int* g_in, int* g_out, unsigned int const n)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    // convert global data pointer to the local pointer of this block
    int* in = g_in + blockDim.x * blockIdx.x * 8;
    
    // unrolling 4 data blocks
    if (idx + blockDim.x * 7 < n) {
        int sum = 0;
        sum += g_in[idx + blockDim.x];
        sum += g_in[idx + blockDim.x * 2];
        sum += g_in[idx + blockDim.x * 3];
        sum += g_in[idx + blockDim.x * 4];
        sum += g_in[idx + blockDim.x * 5];
        sum += g_in[idx + blockDim.x * 6];
        sum += g_in[idx + blockDim.x * 7];
        g_in[idx] += sum;
    }
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            in[tid] += in[tid + stride];
        }

        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0)
        g_out[blockIdx.x] = in[0];
}

// unrolling warps 8
__global__
void reduceUnrollingWarps8(int* g_in, int* g_out, unsigned int const n)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    // convert global data pointer to the local pointer of this block
    int* in = g_in + blockDim.x * blockIdx.x * 8;
    
    // unrolling 4 data blocks
    if (idx + blockDim.x * 7 < n) {
        int sum = 0;
        sum += g_in[idx + blockDim.x];
        sum += g_in[idx + blockDim.x * 2];
        sum += g_in[idx + blockDim.x * 3];
        sum += g_in[idx + blockDim.x * 4];
        sum += g_in[idx + blockDim.x * 5];
        sum += g_in[idx + blockDim.x * 6];
        sum += g_in[idx + blockDim.x * 7];
        g_in[idx] += sum;
    }
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            in[tid] += in[tid + stride];
        }

        __syncthreads();
    }

    // unrolling warp
    if (tid < 32) {
        volatile int* vmem = in;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    // write result for this block to global memory
    if (tid == 0)
        g_out[blockIdx.x] = in[0];
}

// complete unroll warps 8
__global__
void reduceCompleteUnrollWarps8(int* g_in, int* g_out, unsigned int const n)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    // convert global data pointer to the local pointer of this block
    int* in = g_in + blockDim.x * blockIdx.x * 8;
    
    // unrolling 4 data blocks
    if (idx + blockDim.x * 7 < n) {
        int sum = 0;
        sum += g_in[idx + blockDim.x];
        sum += g_in[idx + blockDim.x * 2];
        sum += g_in[idx + blockDim.x * 3];
        sum += g_in[idx + blockDim.x * 4];
        sum += g_in[idx + blockDim.x * 5];
        sum += g_in[idx + blockDim.x * 6];
        sum += g_in[idx + blockDim.x * 7];
        g_in[idx] += sum;
    }
    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512)
        in[tid] += in[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256)
        in[tid] += in[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128)
        in[tid] += in[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64)
        in[tid] += in[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        volatile int* vmem = in;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    // write result for this block to global memory
    if (tid == 0)
        g_out[blockIdx.x] = in[0];
}

// template reduce unrolling warps 8
template<unsigned int BlockSize>
__global__
void reduceCompleteUnroll(int* g_in, int* g_out, unsigned int const n)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    // convert global data pointer to the local pointer of this block
    int* in = g_in + blockDim.x * blockIdx.x * 8;
    
    // unrolling 4 data blocks
    if (idx + blockDim.x * 7 < n) {
        int sum = 0;
        sum += g_in[idx + blockDim.x];
        sum += g_in[idx + blockDim.x * 2];
        sum += g_in[idx + blockDim.x * 3];
        sum += g_in[idx + blockDim.x * 4];
        sum += g_in[idx + blockDim.x * 5];
        sum += g_in[idx + blockDim.x * 6];
        sum += g_in[idx + blockDim.x * 7];
        g_in[idx] += sum;
    }
    __syncthreads();

    // in-place reduction and complete unroll
    if (BlockSize >= 1024 && tid < 512)
        in[tid] += in[tid + 512];
    __syncthreads();

    if (BlockSize >= 512 && tid < 256)
        in[tid] += in[tid + 256];
    __syncthreads();

    if (BlockSize >= 256 && tid < 128)
        in[tid] += in[tid + 128];
    __syncthreads();

    if (BlockSize >= 128 && tid < 64)
        in[tid] += in[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        volatile int* vmem = in;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    // write result for this block to global memory
    if (tid == 0)
        g_out[blockIdx.x] = in[0];
}

int main(int argc, char** argv)
{
    // setup device
    int dev = 0;
    hipDeviceProp_t dev_prop;
    CUDA_ERROR_CHECK(hipGetDeviceProperties(&dev_prop, dev));
    printf("> Starting reduction at device %d: %s\n", dev, dev_prop.name);
    CUDA_ERROR_CHECK(hipSetDevice(dev));

    // array size
    int num_elements = 1 << 24;
    printf("> Array size: %d\n", num_elements);

    // execution configuration
    int block_size = 512;
    if (argc > 1) block_size = atoi(argv[1]);

    dim3 block(block_size);
    dim3 grid((num_elements + block.x - 1) / block.x);
    printf("> grid %d  block %d\n", grid.x, block.x);

    // allocate host memory
    size_t bytes = num_elements * sizeof(int);
    int* h_in = (int*)malloc(bytes);
    int* h_out = (int*)malloc(grid.x * sizeof(int));
    int* tmp = (int*)malloc(bytes);

    // init the input array
    for (int i = 0; i < num_elements; i++)
        h_in[i] = (int)(rand() & 0xFF);
    (void*)memcpy(tmp, h_in, bytes);

    // cuda event
    hipEvent_t start, end;
    CUDA_ERROR_CHECK(hipEventCreate(&start));
    CUDA_ERROR_CHECK(hipEventCreate(&end));

    // allocate device memory
    int *d_in, *d_out;
    CUDA_ERROR_CHECK(hipMalloc(&d_in, bytes));
    CUDA_ERROR_CHECK(hipMalloc(&d_out, grid.x * sizeof(int)));

    float msec = 0.f;
    int gpu_sum = 0;
    
    // cpu reduction
    CUDA_ERROR_CHECK(hipEventRecord(start));
    int cpu_sum = recursiveReduce(tmp, num_elements);
    CUDA_ERROR_CHECK(hipEventRecord(end));
    CUDA_ERROR_CHECK(hipEventSynchronize(end));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));
    printf("cpu reduce                      elapsed %.4f ms    cpu sum: %d\n", msec, cpu_sum);

    // kernel 1: reduceNeighbored
    CUDA_ERROR_CHECK(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipEventRecord(start));
    reduceNeighbored<<<grid, block>>>(d_in, d_out, num_elements);
    CUDA_ERROR_CHECK(hipEventRecord(end));
    CUDA_ERROR_CHECK(hipEventSynchronize(end));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));

    CUDA_ERROR_CHECK(hipMemcpy(h_out, d_out, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += h_out[i];
    printf("gpu Neighbored                  elapsed %.4f ms     gpu sum: %d <<<grid %d block %d>>>\n", msec, gpu_sum, grid.x, block.x);

    // kernel 2: reduceNeighboredLess
    CUDA_ERROR_CHECK(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipEventRecord(start));
    reduceNeighboredLess<<<grid, block>>>(d_in, d_out, num_elements);
    CUDA_ERROR_CHECK(hipEventRecord(end));
    CUDA_ERROR_CHECK(hipEventSynchronize(end));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));

    CUDA_ERROR_CHECK(hipMemcpy(h_out, d_out, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += h_out[i];
    printf("gpu NeighboredLess              elapsed %.4f ms     gpu sum: %d <<<grid %d block %d>>>\n", msec, gpu_sum, grid.x, block.x);

    // kernel 3: reduceInterleaved
    CUDA_ERROR_CHECK(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipEventRecord(start));
    reduceInterleaved<<<grid, block>>>(d_in, d_out, num_elements);
    CUDA_ERROR_CHECK(hipEventRecord(end));
    CUDA_ERROR_CHECK(hipEventSynchronize(end));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));

    CUDA_ERROR_CHECK(hipMemcpy(h_out, d_out, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += h_out[i];
    printf("gpu reduceInterleaved           elapsed %.4f ms     gpu sum: %d <<<grid %d block %d>>>\n", msec, gpu_sum, grid.x, block.x);

    // kernel 4: reduceUnrolling2
    CUDA_ERROR_CHECK(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipEventRecord(start));
    reduceUnrolling2<<<grid.x / 2, block>>>(d_in, d_out, num_elements);
    CUDA_ERROR_CHECK(hipEventRecord(end));
    CUDA_ERROR_CHECK(hipEventSynchronize(end));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));

    CUDA_ERROR_CHECK(hipMemcpy(h_out, d_out, grid.x / 2 * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 2; i++)
        gpu_sum += h_out[i];
    printf("gpu reduceUnrolling2            elapsed %.4f ms     gpu sum: %d <<<grid %d block %d>>>\n", msec, gpu_sum, grid.x / 2, block.x);

    // kernel 5: reduceUnrolling4
    CUDA_ERROR_CHECK(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipEventRecord(start));
    reduceUnrolling4<<<grid.x / 4, block>>>(d_in, d_out, num_elements);
    CUDA_ERROR_CHECK(hipEventRecord(end));
    CUDA_ERROR_CHECK(hipEventSynchronize(end));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));

    CUDA_ERROR_CHECK(hipMemcpy(h_out, d_out, grid.x / 4 * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 4; i++)
        gpu_sum += h_out[i];
    printf("gpu reduceUnrolling4            elapsed %.4f ms     gpu sum: %d <<<grid %d block %d>>>\n", msec, gpu_sum, grid.x / 4, block.x);

    // kernel 6: reduceUnrolling8
    CUDA_ERROR_CHECK(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipEventRecord(start));
    reduceUnrolling8<<<grid.x / 8, block>>>(d_in, d_out, num_elements);
    CUDA_ERROR_CHECK(hipEventRecord(end));
    CUDA_ERROR_CHECK(hipEventSynchronize(end));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));

    CUDA_ERROR_CHECK(hipMemcpy(h_out, d_out, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 8; i++)
        gpu_sum += h_out[i];
    printf("gpu reduceUnrolling8            elapsed %.4f ms     gpu sum: %d <<<grid %d block %d>>>\n", msec, gpu_sum, grid.x / 8, block.x);

    // kernel 7: reduceUnrollingWarps8
    CUDA_ERROR_CHECK(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipEventRecord(start));
    reduceUnrollingWarps8<<<grid.x / 8, block>>>(d_in, d_out, num_elements);
    CUDA_ERROR_CHECK(hipEventRecord(end));
    CUDA_ERROR_CHECK(hipEventSynchronize(end));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));

    CUDA_ERROR_CHECK(hipMemcpy(h_out, d_out, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 8; i++)
        gpu_sum += h_out[i];
    printf("gpu reduceUnrollingWarps8       elapsed %.4f ms     gpu sum: %d <<<grid %d block %d>>>\n", msec, gpu_sum, grid.x / 8, block.x);

    // kernel 8: reduceCompleteUnrollWarps8
    CUDA_ERROR_CHECK(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipEventRecord(start));
    reduceCompleteUnrollWarps8<<<grid.x / 8, block>>>(d_in, d_out, num_elements);
    CUDA_ERROR_CHECK(hipEventRecord(end));
    CUDA_ERROR_CHECK(hipEventSynchronize(end));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));

    CUDA_ERROR_CHECK(hipMemcpy(h_out, d_out, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 8; i++)
        gpu_sum += h_out[i];
    printf("gpu reduceCompleteUnrollWarps8  elapsed %.4f ms     gpu sum: %d <<<grid %d block %d>>>\n", msec, gpu_sum, grid.x / 8, block.x);

    // kernel 9: reduceCompleteUnroll
    CUDA_ERROR_CHECK(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipEventRecord(start));
    switch (block_size) {
        case 1024:
        reduceCompleteUnroll<1024><<<grid.x / 8, block>>>(d_in, d_out, num_elements);
            break;
        case 512:
        reduceCompleteUnroll<512><<<grid.x / 8, block>>>(d_in, d_out, num_elements);
            break;
        case 256:
        reduceCompleteUnroll<256><<<grid.x / 8, block>>>(d_in, d_out, num_elements);
            break;
        case 128:
        reduceCompleteUnroll<128><<<grid.x / 8, block>>>(d_in, d_out, num_elements);
            break;
        case 64:
        reduceCompleteUnroll<64><<<grid.x / 8, block>>>(d_in, d_out, num_elements);
            break;
    }
    CUDA_ERROR_CHECK(hipEventRecord(end));
    CUDA_ERROR_CHECK(hipEventSynchronize(end));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, end));

    CUDA_ERROR_CHECK(hipMemcpy(h_out, d_out, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 8; i++)
        gpu_sum += h_out[i];
    printf("gpu reduceCompleteUnroll        elapsed %.4f ms     gpu sum: %d <<<grid %d block %d>>>\n", msec, gpu_sum, grid.x / 8, block.x);


    // free host memory
    free(h_in);
    free(h_out);
    free(tmp);

    // free device memory
    CUDA_ERROR_CHECK(hipFree(d_in));
    CUDA_ERROR_CHECK(hipFree(d_out));
    CUDA_ERROR_CHECK(hipEventDestroy(start));
    CUDA_ERROR_CHECK(hipEventDestroy(end));

    // reset device
    CUDA_ERROR_CHECK(hipDeviceReset());

    return 0;
}