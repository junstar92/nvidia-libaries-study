#include <iostream>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

__global__
void ldmatrix_test_kernel(half* dst, half const* src, int const count)
{
    __shared__ half shmem[64];

    int tid = threadIdx.x;

    if (2 * tid < count) {
        shmem[2 * tid] = src[2 * tid];
        shmem[2 * tid + 1] = src[2 * tid + 1];
    }
    __syncthreads();

    half* shmem_ptr = nullptr;
    half frag[2];
    if (tid < 8) {
        shmem_ptr = shmem + 8 * tid;
    }

    uint32_t* reg_ptr = reinterpret_cast<uint32_t*>(frag);
    uint32_t shmem_ptr_val = __cvta_generic_to_shared(shmem_ptr);

    asm volatile(
        "ldmatrix.sync.aligned.m8n8.x1.shared.b16 {%0}, [%1];"
        : "=r"(reg_ptr[0])
        : "r"(shmem_ptr_val)
    );

    // printf("[%2d] %f %f\n", tid, __half2float(frag[0]), __half2float(frag[1]));

    if (2 * tid < count) {
        dst[2 * tid] = frag[0];
        dst[2 * tid + 1] = frag[1];
    }
}

int main()
{
    size_t count = 64;
    std::vector<half> h_src, h_dst;

    h_src.resize(count);
    h_dst.resize(count);
    for (size_t i = 0; i < count; i++) {
        h_src[i] = half(static_cast<float>(i));
    }
    std::cout << "Source Matrix:\n";
    for (size_t i = 0; i < count; i++) {
        std::cout << std::setw(5) << std::fixed << std::setprecision(1) << static_cast<float>(h_src[i]) << " ";
        if ((i + 1) % 8 == 0) {
            std::cout << "\n";
        }
    }

    half *d_src_f16, *d_dst_f16;
    hipMalloc(&d_src_f16, count * sizeof(half));
    hipMalloc(&d_dst_f16, count * sizeof(half));

    hipMemcpy(d_src_f16, h_src.data(), count * sizeof(half), hipMemcpyHostToDevice);

    ldmatrix_test_kernel<<<1, 32>>>(d_dst_f16, d_src_f16, count);

    hipMemcpy(h_dst.data(), d_dst_f16, count * sizeof(half), hipMemcpyDeviceToHost);

    std::cout << "\nResult Matrix:\n";
    for (size_t i = 0; i < count; i++) {
        std::cout << std::setw(5) << std::fixed << std::setprecision(1) << float(h_dst[i]) << " ";
        if ((i + 1) % 8 == 0) {
            std::cout << "\n";
        }
    }

    hipFree(d_src_f16);
    hipFree(d_dst_f16);
}