/*****************************************************************************
 * File:        test_SoA.cu
 * Description: This code is for studying the impact on performance of
 *              data layout(SoA) on the GPU.
 * 
 *              To check the performance of data layout, use 'nsight compute'
 *              with 'smsp__sass_average_data_bytes_per_sector_mem_global_op_ld.pct' and
 *              'smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct` metrics.
 *              The first metrics shows the global memory load efficiency and
 *              the second metrics shows the global memory store efficiency.
 * 
 * Compile:     nvcc -O3 -o test_SoA test_SoA.cu
 * Run:         ./test_SoA
 *****************************************************************************/
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_ERROR_CHECK(err) \
    if (err != hipError_t::hipSuccess) { \
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__); \
        exit(err); \
    }

#define NUM_ELEMENTS (1 << 24)

struct innerArray {
    float x[NUM_ELEMENTS];
    float y[NUM_ELEMENTS];
};

void initInnerArray(innerArray* data, int const n)
{
    for (int i = 0; i < n; i++) {
        data->x[i] = static_cast<float>((rand() & 0xFF) / 100.f);
        data->y[i] = static_cast<float>((rand() & 0xFF) / 100.f);
    }
}

__global__
void testInnerArray(innerArray* data, innerArray* result, int const n)
{
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        float tmpx = data->x[i];
        float tmpy = data->y[i];

        tmpx += 10.f;
        tmpy += 20.f;
        result->x[i] = tmpx;
        result->y[i] = tmpy;
    }
}

int main(int argc, char** argv)
{
    // setup device
    int dev = 0;
    hipDeviceProp_t prop;
    CUDA_ERROR_CHECK(hipGetDeviceProperties(&prop, dev));
    printf("> %s: test struct of array at device %d: %s\n", argv[0], dev, prop.name);

    // allocate host memory
    size_t bytes = sizeof(innerArray);
    innerArray* h_A = (innerArray*)malloc(bytes);

    // initialize host array
    initInnerArray(h_A, NUM_ELEMENTS);

    // allocate device memory
    innerArray *d_A, *d_B;
    CUDA_ERROR_CHECK(hipMalloc(&d_A, bytes));
    CUDA_ERROR_CHECK(hipMalloc(&d_B, bytes));

    // copy data from host to device
    CUDA_ERROR_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));

    // execution configuration
    int blocksize = 256;
    if (argc > 1) blocksize = atoi(argv[1]);
    dim3 block(blocksize);
    dim3 grid((NUM_ELEMENTS + block.x - 1) / block.x);

    // CUDA event for estimating elapsed time
    hipEvent_t start, stop;
    CUDA_ERROR_CHECK(hipEventCreate(&start));
    CUDA_ERROR_CHECK(hipEventCreate(&stop));
    float msec;

    // kernel
    CUDA_ERROR_CHECK(hipEventRecord(start));
    testInnerArray<<<grid, block>>>(d_A, d_B, NUM_ELEMENTS);
    CUDA_ERROR_CHECK(hipEventRecord(stop));
    CUDA_ERROR_CHECK(hipEventSynchronize(stop));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&msec, start, stop));
    printf("testInnerArray <<< %3d, %3d >>> elapsed %f ms\n", grid.x, block.x, msec);

    // free memory
    CUDA_ERROR_CHECK(hipFree(d_A));
    CUDA_ERROR_CHECK(hipFree(d_B));
    free(h_A);

    CUDA_ERROR_CHECK(hipDeviceReset());
    return 0;
}